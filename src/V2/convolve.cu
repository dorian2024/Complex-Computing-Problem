#include "hip/hip_runtime.h"
/*********************************************************************
 * convolve.c
 *********************************************************************/

/* Standard includes */
#include <assert.h>
#include <math.h>
#include <stdlib.h>   /* malloc(), realloc() */

/* Our includes */
#include "base.h"
#include "error.h"
#include "convolve.h"
#include "klt_util.h"   /* printing */

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


#define MAX_KERNEL_WIDTH 	71

typedef struct  {
  int width;
  float data[MAX_KERNEL_WIDTH];
}  ConvolutionKernel;

/* Kernels */
static ConvolutionKernel gauss_kernel;
static ConvolutionKernel gaussderiv_kernel;
static float sigma_last = -10.0;


/*********************************************************************
 * _KLTToFloatImage
 *
 * Given a pointer to image data (probably unsigned chars), copy
 * data to a float image.
 */

void _KLTToFloatImage(
  KLT_PixelType *img,
  int ncols, int nrows,
  _KLT_FloatImage floatimg)
{
  KLT_PixelType *ptrend = img + ncols*nrows;
  float *ptrout = floatimg->data;

  /* Output image must be large enough to hold result */
  assert(floatimg->ncols >= ncols);
  assert(floatimg->nrows >= nrows);

  floatimg->ncols = ncols;
  floatimg->nrows = nrows;

  while (img < ptrend)  *ptrout++ = (float) *img++;
}


/*********************************************************************
 * _computeKernels
 */

static void _computeKernels(
  float sigma,
  ConvolutionKernel *gauss,
  ConvolutionKernel *gaussderiv)
{
  const float factor = 0.01f;   /* for truncating tail */
  int i;

  assert(MAX_KERNEL_WIDTH % 2 == 1);
  assert(sigma >= 0.0);

  /* Compute kernels, and automatically determine widths */
  {
    const int hw = MAX_KERNEL_WIDTH / 2;
    float max_gauss = 1.0f, max_gaussderiv = (float) (sigma*exp(-0.5f));
	
    /* Compute gauss and deriv */
    for (i = -hw ; i <= hw ; i++)  {
      gauss->data[i+hw]      = (float) exp(-i*i / (2*sigma*sigma));
      gaussderiv->data[i+hw] = -i * gauss->data[i+hw];
    }

    /* Compute widths */
    gauss->width = MAX_KERNEL_WIDTH;
    for (i = -hw ; fabs(gauss->data[i+hw] / max_gauss) < factor ; 
         i++, gauss->width -= 2);
    gaussderiv->width = MAX_KERNEL_WIDTH;
    for (i = -hw ; fabs(gaussderiv->data[i+hw] / max_gaussderiv) < factor ; 
         i++, gaussderiv->width -= 2);
    if (gauss->width == MAX_KERNEL_WIDTH || 
        gaussderiv->width == MAX_KERNEL_WIDTH)
      KLTError("(_computeKernels) MAX_KERNEL_WIDTH %d is too small for "
               "a sigma of %f", MAX_KERNEL_WIDTH, sigma);
  }

  /* Shift if width less than MAX_KERNEL_WIDTH */
  for (i = 0 ; i < gauss->width ; i++)
    gauss->data[i] = gauss->data[i+(MAX_KERNEL_WIDTH-gauss->width)/2];
  for (i = 0 ; i < gaussderiv->width ; i++)
    gaussderiv->data[i] = gaussderiv->data[i+(MAX_KERNEL_WIDTH-gaussderiv->width)/2];
  /* Normalize gauss and deriv */
  {
    const int hw = gaussderiv->width / 2;
    float den;
			
    den = 0.0;
    for (i = 0 ; i < gauss->width ; i++)  den += gauss->data[i];
    for (i = 0 ; i < gauss->width ; i++)  gauss->data[i] /= den;
    den = 0.0;
    for (i = -hw ; i <= hw ; i++)  den -= i*gaussderiv->data[i+hw];
    for (i = -hw ; i <= hw ; i++)  gaussderiv->data[i+hw] /= den;
  }

  sigma_last = sigma;
}
	

/*********************************************************************
 * _KLTGetKernelWidths
 *
 */

void _KLTGetKernelWidths(
  float sigma,
  int *gauss_width,
  int *gaussderiv_width)
{
  _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);
  *gauss_width = gauss_kernel.width;
  *gaussderiv_width = gaussderiv_kernel.width;
}


/*********************************************************************
 * _convolveImageHoriz
 */

//this function it blurs or filters the image along the x-direction (left to right) using a 1D kernel.
//static void _convolveImageHoriz(
//  _KLT_FloatImage imgin, 
//  ConvolutionKernel kernel,
//  _KLT_FloatImage imgout)
//{
//  float *ptrrow = imgin->data;           /* Points to row's first pixel */
//  register float *ptrout = imgout->data, /* Points to next output pixel */
//    *ppp;
//  register float sum;
//  register int radius = kernel.width / 2;
//  register int ncols = imgin->ncols, nrows = imgin->nrows;
//  register int i, j, k;
//
//  /* Kernel width must be odd */
//  assert(kernel.width % 2 == 1);
//
//  /* Must read from and write to different images */
//  assert(imgin != imgout);
//
//  /* Output image must be large enough to hold result */
//  assert(imgout->ncols >= imgin->ncols);
//  assert(imgout->nrows >= imgin->nrows);
//
//  /* For each row, do ... */
//  for (j = 0 ; j < nrows ; j++)  {
//
//    /* Zero leftmost columns */
//    for (i = 0 ; i < radius ; i++)
//      *ptrout++ = 0.0;
//
//    /* Convolve middle columns with kernel */
//    for ( ; i < ncols - radius ; i++)  {
//      ppp = ptrrow + i - radius;
//      sum = 0.0;
//      for (k = kernel.width-1 ; k >= 0 ; k--)
//        sum += *ppp++ * kernel.data[k];
//      *ptrout++ = sum;
//    }
//
//    /* Zero rightmost columns */
//    for ( ; i < ncols ; i++)
//      *ptrout++ = 0.0;
//
//    ptrrow += ncols;
//  }
//}

__global__ void convolveImageHorizKernel(
    const float* imgin,
    const float* kernel_data,
    int kernel_width,
    int ncols,
    int nrows,
    float* imgout)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // column index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // row index

    if (i >= ncols || j >= nrows)
        return;

    int radius = kernel_width / 2;
    int out_idx = j * ncols + i;

    // Zero leftmost columns
    if (i < radius) {
        imgout[out_idx] = 0.0f;
        return;
    }

    // Zero rightmost columns
    if (i >= ncols - radius) {
        imgout[out_idx] = 0.0f;
        return;
    }

    // Convolve middle columns with kernel
    float sum = 0.0f;
    int row_start = j * ncols;

    for (int k = kernel_width - 1; k >= 0; k--) {
        int ppp_idx = row_start + i - radius + (kernel_width - 1 - k);
        sum += imgin[ppp_idx] * kernel_data[k];
    }

    imgout[out_idx] = sum;
}

// CUDA version of _convolveImageHoriz
static void _convolveImageHoriz_cuda(
    _KLT_FloatImage imgin,
    ConvolutionKernel kernel,
    _KLT_FloatImage imgout)
{
    int ncols = imgin->ncols;
    int nrows = imgin->nrows;
    size_t img_size = ncols * nrows * sizeof(float);
    size_t kernel_size = kernel.width * sizeof(float);

    // Allocate device memory
    float* d_imgin, * d_imgout, * d_kernel;
    hipMalloc(&d_imgin, img_size);
    hipMalloc(&d_imgout, img_size);
    hipMalloc(&d_kernel, kernel_size);

    // Copy data to device
    hipMemcpy(d_imgin, imgin->data, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data, kernel_size, hipMemcpyHostToDevice);

    // Configure and launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((ncols + blockDim.x - 1) / blockDim.x,
        (nrows + blockDim.y - 1) / blockDim.y);

    convolveImageHorizKernel << <gridDim, blockDim >> > (
        d_imgin, d_kernel, kernel.width, ncols, nrows, d_imgout);

    // Copy result back to host
    hipMemcpy(imgout->data, d_imgout, img_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_imgin);
    hipFree(d_imgout);
    hipFree(d_kernel);
}

/*********************************************************************
 * _convolveImageVert
 */

//static void _convolveImageVert(
//  _KLT_FloatImage imgin,
//  ConvolutionKernel kernel,
//  _KLT_FloatImage imgout)
//{
//  float *ptrcol = imgin->data;            /* Points to row's first pixel */
//  register float *ptrout = imgout->data,  /* Points to next output pixel */
//    *ppp;
//  register float sum;
//  register int radius = kernel.width / 2;
//  register int ncols = imgin->ncols, nrows = imgin->nrows;
//  register int i, j, k;
//
//  /* Kernel width must be odd */
//  assert(kernel.width % 2 == 1);
//
//  /* Must read from and write to different images */
//  assert(imgin != imgout);
//
//  /* Output image must be large enough to hold result */
//  assert(imgout->ncols >= imgin->ncols);
//  assert(imgout->nrows >= imgin->nrows);
//
//  /* For each column, do ... */
//  for (i = 0 ; i < ncols ; i++)  {
//
//    /* Zero topmost rows */
//    for (j = 0 ; j < radius ; j++)  {
//      *ptrout = 0.0;
//      ptrout += ncols;
//    }
//
//    /* Convolve middle rows with kernel */
//    for ( ; j < nrows - radius ; j++)  {
//      ppp = ptrcol + ncols * (j - radius);
//      sum = 0.0;
//      for (k = kernel.width-1 ; k >= 0 ; k--)  {
//        sum += *ppp * kernel.data[k];
//        ppp += ncols;
//      }
//      *ptrout = sum;
//      ptrout += ncols;
//    }
//
//    /* Zero bottommost rows */
//    for ( ; j < nrows ; j++)  {
//      *ptrout = 0.0;
//      ptrout += ncols;
//    }
//
//    ptrcol++;
//    ptrout -= nrows * ncols - 1;
//  }
//}

// CUDA kernel for vertical convolution
__global__ void convolveImageVertKernel(
    const float* imgin,
    const float* kernel_data,
    int kernel_width,
    int ncols,
    int nrows,
    float* imgout)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // column index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // row index
    
    if (i >= ncols || j >= nrows)
        return;
    
    int radius = kernel_width / 2;
    int out_idx = j * ncols + i;
    
    // Zero topmost rows
    if (j < radius) {
        imgout[out_idx] = 0.0f;
        return;
    }
    
    // Zero bottommost rows
    if (j >= nrows - radius) {
        imgout[out_idx] = 0.0f;
        return;
    }
    
    // Convolve middle rows with kernel
    float sum = 0.0f;
    
    for (int k = kernel_width - 1; k >= 0; k--) {
        int ppp_idx = (j - radius + (kernel_width - 1 - k)) * ncols + i;
        sum += imgin[ppp_idx] * kernel_data[k];
    }
    
    imgout[out_idx] = sum;
}

// CUDA version of _convolveImageVert
static void _convolveImageVert_cuda(
    _KLT_FloatImage imgin,
    ConvolutionKernel kernel,
    _KLT_FloatImage imgout)
{
    int ncols = imgin->ncols;
    int nrows = imgin->nrows;
    size_t img_size = ncols * nrows * sizeof(float);
    size_t kernel_size = kernel.width * sizeof(float);

    // Allocate device memory
    float* d_imgin, * d_imgout, * d_kernel;
    hipMalloc(&d_imgin, img_size);
    hipMalloc(&d_imgout, img_size);
    hipMalloc(&d_kernel, kernel_size);

    // Copy data to device
    hipMemcpy(d_imgin, imgin->data, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data, kernel_size, hipMemcpyHostToDevice);

    // Configure and launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((ncols + blockDim.x - 1) / blockDim.x,
        (nrows + blockDim.y - 1) / blockDim.y);

    convolveImageVertKernel << <gridDim, blockDim >> > (
        d_imgin, d_kernel, kernel.width, ncols, nrows, d_imgout);

    // Copy result back to host
    hipMemcpy(imgout->data, d_imgout, img_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_imgin);
    hipFree(d_imgout);
    hipFree(d_kernel);
}


/*********************************************************************
 * _convolveSeparate
 */

static void _convolveSeparate(
    _KLT_FloatImage imgin,
    ConvolutionKernel horiz_kernel,
    ConvolutionKernel vert_kernel,
    _KLT_FloatImage imgout)
{
    /* Create temporary image */
    _KLT_FloatImage tmpimg;
    tmpimg = _KLTCreateFloatImage(imgin->ncols, imgin->nrows);
    
    /* Do convolution */
    _convolveImageHoriz_cuda(imgin, horiz_kernel, tmpimg);
    _convolveImageVert_cuda(tmpimg, vert_kernel, imgout);
    
    /* Free memory */
    _KLTFreeFloatImage(tmpimg);
}

	
/*********************************************************************
 * _KLTComputeGradients
 */

void _KLTComputeGradients(
  _KLT_FloatImage img,
  float sigma,
  _KLT_FloatImage gradx,
  _KLT_FloatImage grady)
{
				
  /* Output images must be large enough to hold result */
  assert(gradx->ncols >= img->ncols);
  assert(gradx->nrows >= img->nrows);
  assert(grady->ncols >= img->ncols);
  assert(grady->nrows >= img->nrows);

  /* Compute kernels, if necessary */
  if (fabs(sigma - sigma_last) > 0.05)
    _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);
	
  _convolveSeparate(img, gaussderiv_kernel, gauss_kernel, gradx);
  _convolveSeparate(img, gauss_kernel, gaussderiv_kernel, grady);

}
	

/*********************************************************************
 * _KLTComputeSmoothedImage
 */

void _KLTComputeSmoothedImage(
  _KLT_FloatImage img,
  float sigma,
  _KLT_FloatImage smooth)
{
  /* Output image must be large enough to hold result */
  assert(smooth->ncols >= img->ncols);
  assert(smooth->nrows >= img->nrows);

  /* Compute kernel, if necessary; gauss_deriv is not used */
  if (fabs(sigma - sigma_last) > 0.05)
    _computeKernels(sigma, &gauss_kernel, &gaussderiv_kernel);

  _convolveSeparate(img, gauss_kernel, gauss_kernel, smooth);
}


